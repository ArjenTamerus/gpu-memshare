
#include <hip/hip_runtime.h>
#include <iostream>

#define N_MEMELEM 32

__global__ void init_devmem(double *mem)
{
	mem[blockIdx.x*blockDim.x + threadIdx.x] = blockIdx.x*blockDim.x + threadIdx.x;
}

__global__ void double_devmem(double *mem)
{
	mem[blockIdx.x*blockDim.x + threadIdx.x] *= 2;
}

void do_cuda_init(double *dev_mem)
{
	init_devmem<<<N_MEMELEM/32,32>>>(dev_mem);
}

void do_cuda_double(double *dev_mem)
{
	double_devmem<<<N_MEMELEM/32,32>>>(dev_mem);
}

hipIpcMemHandle_t get_memhandle(void *devmem)
{
	hipError_t err_dev;
	hipIpcMemHandle_t dev_mem_handle;

	err_dev = hipIpcGetMemHandle(&dev_mem_handle, devmem);
	std::cout << "getHandle: " << hipGetErrorName(err_dev) << ": " << hipGetErrorString(err_dev) << std::endl;

	return dev_mem_handle;
}

void *cuda_open_handle(hipIpcMemHandle_t dev_mem_handle)
{
	hipError_t err_dev;
	void *mem_ptr=NULL;

	err_dev = hipIpcOpenMemHandle(&mem_ptr, dev_mem_handle, hipIpcMemLazyEnablePeerAccess);
	std::cout << "openHandle: " << hipGetErrorName(err_dev) << ": " << hipGetErrorString(err_dev) << std::endl;

	return mem_ptr;
}

void cuda_close_handle(void *devmem)
{
	hipError_t err_dev;

	err_dev = hipIpcCloseMemHandle(devmem);
	std::cout << "closeHandle: " << hipGetErrorName(err_dev) << ": " << hipGetErrorString(err_dev) << std::endl;
}

